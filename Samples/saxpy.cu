#include "hip/hip_runtime.h"
#include "../Framework/lskel.cuh"
#include <iostream>

struct Saxpy{
    float a;
    Saxpy(float a_) : a(a_) {}

    template<typename T1, typename T2>
    __host__ __device__ float operator()(const T1& x, const T2& y) const {
        return a * x + y;
    }
};

int main() {
    const size_t N = 1 << 20;
    const float a = 2.0f;

    lskel::Vector<float> x(N);
    lskel::Vector<float> y(N);

    // Initialize x and y with a sequence of values
    x.fill_with_sequence(); // Fills x with 1.0, 2.0, 3.0, ...
    y.fill_with_sequence(); // Fills y with 1.0, 2.0, 3.0, ...

    // Perform SAXPY operation: y = a * x + y

    y = y.map(x, Saxpy(a));

    // Verify the result
    bool correct = true;
    for (size_t i = 0; i < N; ++i) {
        float expected = a * (i + 1.0f) + (i + 1.0f);
        if (fabs(y[i] - expected) > 1e-5) {
            correct = false;
            std::cout << "Error at index " << i << ": expected " << expected << ", got " << y[i] << std::endl;
            break;
        }
    }

    if (correct) {
        std::cout << "SAXPY operation completed successfully." << std::endl;
    }

    return 0;
}
