#include "hip/hip_runtime.h"
#include "../Framework/lskel.cuh"
#include <iostream>

//vector norm functor
struct Square
{
    template<typename T>
    __host__ __device__ T operator()(const T& x) const {
        return x * x;
    }
};

struct Plus
{
    template<typename T>
    __host__ __device__ T operator()(const T& a, const T& b) const {
        return a + b;
    }
};

int main() {
    const size_t N = 1 << 20;

    lskel::Vector<float> x(N);

    // Initialize x with a sequence of values
    x.fill_with_sequence(); // Fills x with 1.0, 2.0, 3.0, ...

    // Compute the norm of the vector
    float sum_of_squares = x.map(Square()).reduce(Plus(), 0.0f);

    float norm = std::sqrt(sum_of_squares);

    x = x / norm;

    // Print the first 10 elements of the normalized vector
    std::cout << "First 10 elements of the normalized vector:\n";
    for (size_t i = 0; i < 10; ++i) {
        std::cout << x[i] << " ";
    }
    
    //Expected
    std::cout << "\nExpected:\n";
    for (size_t i = 1; i <= 10; ++i) {
        std::cout << (i / norm) << " ";
    }

    return 0;
}
